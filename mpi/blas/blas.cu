#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "params.h"

#define BLOCK_SIZE 16
#define A_MAT(x,y,N) A[x * N + y]
#define B_MAT(x,y,N) B[x * N + y]
#define C_MAT(x,y,N) C[x * N + y]

__global__ void matmul(Real* A, Real* B, Real* C, int N) {
  int ti = threadIdx.x;
  int tj = threadIdx.y;

  int ci = BLOCK_SIZE * blockIdx.x + ti;
  int cj = BLOCK_SIZE * blockIdx.y + tj;

  if (ci < N && cj < N) {
    Real C_sub = 0.0f;

    int A_j = tj;
    int B_i = ti;

    __shared__ Real s_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ Real s_B[BLOCK_SIZE][BLOCK_SIZE];

    for (int l = 0; l < (N + BLOCK_SIZE - 1) / BLOCK_SIZE; l++) {
      s_A[ti][tj] = A_MAT(ci, A_j, N);
      s_B[ti][tj] = B_MAT(B_i, cj, N);
      __syncthreads();

      for (int q = 0; q < BLOCK_SIZE; q++) {
        C_sub += s_A[ti][q] * s_B[q][tj];
      }

      A_j += BLOCK_SIZE;
      B_i += BLOCK_SIZE;
      __syncthreads();
    }

    C_MAT(ci, cj, N) = C_sub;
  }
}

void runCuda(Params* params, hipStream_t stream, hipblasHandle_t handle) {
  // Unpack parameters
  Real* h_A = params->h_data[0];
  Real* h_B = params->h_data[1];
  Real* h_C = params->h_data[2];
  Real* d_A = params->d_data[0];
  Real* d_B = params->d_data[1];
  Real* d_C = params->d_data[2];
  int N = params->n_elems;
  size_t size = params->mem_size;

  if (!params->cublas) {
    // Use simple handwritten kernel
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dim_grid(ceil((Real)N / dim_block.x), ceil((Real)N / dim_block.y));

    hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream);

    switch (params->type) {
      case Comp::DOT:
        // TODO
        break;
      case Comp::GEMM:
        matmul<<<dim_grid, dim_block, 0, stream>>>(d_A, d_B, d_C, N);
        hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream);
        break;
    }
  }
  else {
    // Use the cuBLAS library
    switch (params->type) {
      case Comp::DOT:
        // TODO
        break;
      case Comp::GEMM:
        Real alpha = 1.0f;
        Real beta = 0.0f;

        hipblasSetMatrixAsync(N, N, sizeof(Real), h_A, N, d_A, N, stream);
        hipblasSetMatrixAsync(N, N, sizeof(Real), h_B, N, d_B, N, stream);

        // need to switch A and B due to how cuBLAS sees arrays in Fortran style
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N,
            d_A, N, &beta, d_C, N);

        hipblasGetMatrixAsync(N, N, sizeof(Real), d_C, N, h_C, N, stream);
        break;
    }
  }

  hipStreamSynchronize(stream);

  std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
}
