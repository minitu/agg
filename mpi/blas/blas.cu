#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "params.h"
#include "comp.h"

#define N_PER_THREAD 32
#define BLOCK_SIZE 16
#define A_MAT(x,y,N) A[x * N + y]
#define B_MAT(x,y,N) B[x * N + y]
#define C_MAT(x,y,N) C[x * N + y]

__global__ void dotp(Real* A, Real* B, Real* C, Integer N) {
  Integer gi = (BLOCK_SIZE * BLOCK_SIZE) * blockIdx.x + threadIdx.x;
  Integer first_idx = gi * N_PER_THREAD;
  Integer last_idx = (gi + 1) * N_PER_THREAD - 1;

  if (first_idx < N) {
    Real sum = (Real)0.0;
    for (Integer i = first_idx; i <= last_idx && i < N; i++) {
      sum += A[i] * B[i];
    }

    atomicAdd(C, sum);
  }
}

// TODO change int to Integer
__global__ void matmul(Real* A, Real* B, Real* C, int N) {
  int ti = threadIdx.x;
  int tj = threadIdx.y;

  int ci = BLOCK_SIZE * blockIdx.x + ti;
  int cj = BLOCK_SIZE * blockIdx.y + tj;

  if (ci < N && cj < N) {
    Real C_sub = 0.0f;

    int A_j = tj;
    int B_i = ti;

    __shared__ Real s_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ Real s_B[BLOCK_SIZE][BLOCK_SIZE];

    for (int l = 0; l < (N + BLOCK_SIZE - 1) / BLOCK_SIZE; l++) {
      s_A[ti][tj] = A_MAT(ci, A_j, N);
      s_B[ti][tj] = B_MAT(B_i, cj, N);
      __syncthreads();

      for (int q = 0; q < BLOCK_SIZE; q++) {
        C_sub += s_A[ti][q] * s_B[q][tj];
      }

      A_j += BLOCK_SIZE;
      B_i += BLOCK_SIZE;
      __syncthreads();
    }

    C_MAT(ci, cj, N) = C_sub;
  }
}

void runCuda(Comp* comp, Params* params, hipStream_t stream,
    hipblasHandle_t handle, int rank) {
  Real* h_A;
  Real* h_B;
  Real* h_C;
  Real* d_A;
  Real* d_B;
  Real* d_C;
  Integer N;
  Integer size;

  // Unpack Comp
  if (!comp->agg) {
    h_A = comp->h_A;
    h_B = comp->h_B;
    h_C = comp->h_C;
    d_A = comp->d_A;
    d_B = comp->d_B;
    d_C = comp->d_C;
    N = comp->N;
    size = comp->mem_size;
  }
  else {
    h_A = comp->h_GA;
    h_B = comp->h_GB;
    h_C = comp->h_GC;
    d_A = comp->d_GA;
    d_B = comp->d_GB;
    d_C = comp->d_GC;
    N = comp->N * comp->n_ranks;
    size = comp->mem_size * comp->n_ranks;
  }

  if (!params->cublas) {
    // Use simple handwritten kernel
    dim3 dim_block;
    dim3 dim_grid;

    switch (params->type) {
      case CompType::DOT_GLOBAL:
        dim_block = dim3(BLOCK_SIZE * BLOCK_SIZE);
        dim_grid = dim3(ceil((Real)N / (dim_block.x * N_PER_THREAD)));

        hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream);

        dotp<<<dim_grid, dim_block, 0, stream>>>(d_A, d_B, d_C, N);

        hipMemcpyAsync(h_C, d_C, sizeof(Real), hipMemcpyDeviceToHost, stream);

        break;
      // TODO DOT_LOCAL
      case CompType::GEMM_GLOBAL:
      case CompType::GEMM_LOCAL:
        // TODO
        dim_block = dim3(BLOCK_SIZE, BLOCK_SIZE);
        dim_grid = dim3(ceil((Real)N / dim_block.x), ceil((Real)N / dim_block.y));

        hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream);

        matmul<<<dim_grid, dim_block, 0, stream>>>(d_A, d_B, d_C, N);

        hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream);

        break;
    }
  }
  else {
    // Use the cuBLAS library
    switch (params->type) {
      case CompType::DOT_GLOBAL:
      case CompType::DOT_LOCAL:
        hipblasSetVectorAsync(N, sizeof(Real), h_A, 1, d_A, 1, stream);
        hipblasSetVectorAsync(N, sizeof(Real), h_B, 1, d_B, 1, stream);

        hipblasSdot(handle, N, d_A, 1, d_B, 1, h_C);

        break;
      case CompType::GEMM_GLOBAL:
      case CompType::GEMM_LOCAL:
        // TODO
        Real alpha = 1.0f;
        Real beta = 0.0f;

        hipblasSetMatrixAsync(N, N, sizeof(Real), h_A, N, d_A, N, stream);
        hipblasSetMatrixAsync(N, N, sizeof(Real), h_B, N, d_B, N, stream);

        // need to switch A and B due to how cuBLAS sees arrays in Fortran style
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N,
            d_A, N, &beta, d_C, N);

        hipblasGetMatrixAsync(N, N, sizeof(Real), d_C, N, h_C, N, stream);
        break;
    }
  }

  hipStreamSynchronize(stream);

  if (hipPeekAtLastError() != hipSuccess)
    std::cerr << "[MPI " << rank << "] CUDA error: "
      << hipGetErrorString(hipGetLastError()) << std::endl;
}
