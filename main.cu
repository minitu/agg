#include "hip/hip_runtime.h"
#include "main.h"

__global__ void empty() {
}

Main::Main(CkArgMsg* m) {
  int sMemSize = 0;
  int numCores = 5120; // TODO query this
  int numThreads = 64; // TODO query this
  int numExperiments =  1000; // default
  int numStreams     = 1;//0000; // default
  hipStream_t streams[numStreams];

  if (m->argc > 1) {
    numExperiments = atoi(m->argv[1]);
  }

  delete m;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time;

#if 0
  for (int i = 0; i < numStreams; i++) {
    hipStreamCreate(&streams[i]);
  }
#endif

  // Warm Up
  for (int i = 0; i < 100; i++) {
    //empty<<<numCores / numThreads, numThreads>>>();
    empty<<<1,1>>>();
  }

  hipDeviceSynchronize();

  // Should we measure each launch and sum or loop over?
  // - loop

  // Tests
  // sMemSize   - no effect
  // 0 stream   - no effect
  // synchroize - no effect
  // > 1 thread, and block - no effect
  // > 1 thread
  // > 1 block
  // non-default stream

  // GPU Manager
  // With data, without data
  for (int i = 0; i < 1; i++) {
    // 1, 10, 100, 1000, 10000  kernel launch(es)
    /*for (int j = 1; j < 100001; j *= 10)*/ {
      // Need to take averages
#if 0
      hipEventRecord(start);
      hipEventRecord(start, streams[i]);
#else
      hipEventRecord(start, 0);
#endif
      for(int experiment = 0; experiment < numExperiments; experiment++) {
        // How to measure time?
        // events
        // CPU time
        // - clock()
        // - gettimeofday
        // - c++11
        // - clock_gettime
        // - cutTimer (deprecated?)
#if 0
        empty<<<numCores / numThreads, numThreads, sMemSize, streams[i]>>>();
        empty<<<numCores / numThreads, numThreads, sMemSize>>>();
        empty<<<numCores / numThreads, numThreads>>>();
        empty<<<1, 1, sMemSize, 0>>>();
#else
        empty<<<numCores / numThreads, numThreads, sMemSize, 0>>>();
#endif
      }
#if 0
      hipEventRecord(stop);
      hipEventRecord(stop, streams[i]);
#else
      hipEventRecord(stop, 0);
#endif

      //hipDeviceSynchronize();
      hipEventSynchronize(stop);

      hipEventElapsedTime(&time, start, stop);

      CkPrintf("Total   Launch Latency: %.4f\tms\n", time);
      CkPrintf("Average Launch Latency: %.1f\tus\n", time / numExperiments * 1000);
    }
  }

  CkExit();
}

#include "main.def.h"
